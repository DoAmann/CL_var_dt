#include "hip/hip_runtime.h"
//calculate index for complex lattice
__device__ inline int ind(int compindex,int xindex,int yindex, int zindex, int tauindex)
{
	return compindex*TAUSIZE*XSIZE*YSIZE*ZSIZE+my_map(tauindex,TAUSIZE)*XSIZE*YSIZE*ZSIZE+my_map(xindex,XSIZE)*YSIZE*ZSIZE+my_map(yindex,YSIZE)*ZSIZE+my_map(zindex,ZSIZE);
}

__device__ inline int ind_rand(int compindex,int xindex,int yindex, int zindex, int tauindex, int part)
{
	//return compindex*TAUSIZE*XSIZE*YSIZE*ZSIZE*2+tauindex*XSIZE*YSIZE*ZSIZE*2+xindex*YSIZE*ZSIZE*2+yindex*ZSIZE+zindex*2+part;
	return compindex*2*XSIZE*YSIZE*ZSIZE*TAUSIZE+xindex*2*YSIZE*ZSIZE*TAUSIZE+yindex*2*ZSIZE*TAUSIZE+zindex*2*TAUSIZE+tauindex*2+part;
}

__device__ inline int indl(int compindex,int xindex,int yindex, int zindex, int tauindex)
{
	return compindex*(BLOCKSIZETAU+2)*(BLOCKSIZEX+2)*(BLOCKSIZEY+2)*(BLOCKSIZEZ+2)+(tauindex+1)*(BLOCKSIZEX+2)*(BLOCKSIZEY+2)*(BLOCKSIZEZ+2)+(xindex+1)*(BLOCKSIZEY+2)*(BLOCKSIZEZ+2)+(yindex+1)*(BLOCKSIZEZ+2)+zindex+1;
}


//the Laplacian is implemented in a "local" version, taking as coordinates block coordinates, and a "global" version, taking lattice coordinates

__device__ inline hipDoubleComplex laplacian(hipDoubleComplex *fields, int i, int j, int k, int l, int m)
{
	return fields[ind(i,j-1,k,l,m)]+fields[ind(i,j+1,k,l,m)]+fields[ind(i,j,k-1,l,m)]+fields[ind(i,j,k+1,l,m)]+fields[ind(i,j,k,l-1,m)]+fields[ind(i,j,k,l+1,m)]-6.*fields[ind(i,j,k,l,m)];
}

__device__ inline hipDoubleComplex laplacian_l(hipDoubleComplex *fields, int i, int j, int k, int l, int m)
{
	return fields[indl(i,j-1,k,l,m)]+fields[indl(i,j+1,k,l,m)]+fields[indl(i,j,k-1,l,m)]+fields[indl(i,j,k+1,l,m)]+fields[indl(i,j,k,l-1,m)]+fields[indl(i,j,k,l+1,m)]-6.*fields[indl(i,j,k,l,m)];
}


__global__ void summation(hipDoubleComplex *data1, hipDoubleComplex *data2)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	data1[x]=data1[x]+data2[x];
}

__global__ void multiplication(hipDoubleComplex *data1, hipDoubleComplex *data2)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	data1[x]=data1[x]*data2[x];
}

__global__ void normalization(hipDoubleComplex *data, double factor)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	data[x]=factor*data[x];
}

__global__ void set_value(hipDoubleComplex *data, double re_psi, double im_psi, bool conjug, double P, double MU)
{
	const int j=blockDim.x * blockIdx.x + threadIdx.x;
	const int k=blockDim.y * blockIdx.y + threadIdx.y;
	const int l=int((blockDim.z * blockIdx.z + threadIdx.z)/TAUSIZE);
	const int m=int((blockDim.z * blockIdx.z + threadIdx.z)%TAUSIZE);
	
	hipDoubleComplex value;
	if (conjug)
	{
		value=make_hipDoubleComplex(re_psi, -im_psi);
	}
	else
	{
		value=make_hipDoubleComplex(re_psi,im_psi);
	}
	/*
	for(int i=0; i<COMPONENTS; i++)
	{
		data[ind(i,j,k,l,m)]=value;
	}
	*/
	for(int i=0; i<COMPONENTS; i++)
	{	
		if (i==0 || i==2){
		data[ind(i,j,k,l,m)]=value;
		}
		else if (i==1||i==3){
		data[ind(i,j,k,l,m)]=make_hipDoubleComplex(sqrt(2.5/3.0),0.)*value;
		}
	}
}

class ComplexLattice
{
	public:
		ComplexLattice(bool conjug);
		~ComplexLattice();
		void fft();
		void fft_inv();
		void normalize(double factor);
		void set_mean_field(double re_psi, double im_psi, double P, double MU);
		void operator *=(ComplexLattice &other);
		void operator +=(ComplexLattice &other);
		void operator =(ComplexLattice &other);
		hipDoubleComplex *get_pointer();
	private:
		hipDoubleComplex *store;
		int length;
		bool conjugated;
		dim3 dimblock;
		dim3 dimgrid;
		dim3 dimblock2;
		dim3 dimgrid2;
		hipfftHandle plan;
		void make_plan();
};

ComplexLattice::ComplexLattice(bool conjug)
{
	length=COMPONENTS*XSIZE*YSIZE*ZSIZE*TAUSIZE;
	conjugated=conjug;
	hipMalloc(&store,length*sizeof(hipDoubleComplex));
	
	dim3 temp_dimblock(BLOCKSIZEX ,BLOCKSIZEY,BLOCKSIZEZ*BLOCKSIZETAU);
	dim3 temp_dimgrid(int(XSIZE/BLOCKSIZEX), int(YSIZE/BLOCKSIZEY), int(ZSIZE*TAUSIZE/(BLOCKSIZEZ*BLOCKSIZETAU)));
	dimblock=temp_dimblock;
	dimgrid=temp_dimgrid;
	
	dim3 temp_dimblock2(BLOCKSIZE,1,1);
	dim3 temp_dimgrid2(int(length/BLOCKSIZE),1,1);
	dimblock2=temp_dimblock2;
	dimgrid2=temp_dimgrid2;
	
	make_plan();
}

ComplexLattice::~ComplexLattice()
{
	hipFree(store);
	hipfftDestroy(plan);
}



void ComplexLattice::make_plan()
{
	int rank = ZSIZE > 1 ? 3 : ((ZSIZE==1 && YSIZE > 1) ? 2 : ((ZSIZE==1 && YSIZE==1 && XSIZE>1) ? 1 : 0));
	int n[rank];

	if (rank == 1)
	{
		n[0] = XSIZE;
	}
	else if (rank == 2)
	{
		n[0] = XSIZE;
		n[1] = YSIZE;
	}
	else if (rank == 3)
	{
		n[0] = XSIZE;
		n[1] = YSIZE;
		n[2] = ZSIZE;
	}
	hipfftPlanMany(&plan, rank, n, NULL, 0, 0, NULL, 0, 0, HIPFFT_Z2Z, COMPONENTS*TAUSIZE);
}

void ComplexLattice::fft()
{
	if(conjugated)
	{
		hipfftExecZ2Z(plan, store, store, HIPFFT_FORWARD);
	}
	else
	{
		hipfftExecZ2Z(plan, store, store, HIPFFT_BACKWARD);
	}
}

void ComplexLattice::fft_inv()
{
	if(conjugated)
	{
		hipfftExecZ2Z(plan, store, store, HIPFFT_BACKWARD);
	}
	else
	{
		hipfftExecZ2Z(plan, store, store, HIPFFT_FORWARD);
	}
}

void ComplexLattice::normalize(double factor)
{
	hipDeviceSynchronize();
	normalization<<<dimgrid2,dimblock2>>>(store, factor);
	hipDeviceSynchronize();
}


void ComplexLattice::set_mean_field(double re_psi, double im_psi, double P, double MU)
{
	hipDeviceSynchronize();
	set_value<<<dimgrid,dimblock>>>(store,re_psi, im_psi, conjugated, P, MU);
	hipDeviceSynchronize();	
}

void ComplexLattice::operator +=(ComplexLattice &other)
{
	hipDeviceSynchronize();
	summation<<<dimgrid2,dimblock2>>>(store, other.store);
	hipDeviceSynchronize();
}

void ComplexLattice::operator *=(ComplexLattice &other)
{
	hipDeviceSynchronize();
	multiplication<<<dimgrid2,dimblock2>>>(store, other.store);
	hipDeviceSynchronize();
}

void ComplexLattice::operator =(ComplexLattice &other)
{
	hipMemcpy(store,other.store,length*sizeof(hipDoubleComplex) ,hipMemcpyDeviceToDevice);
}


hipDoubleComplex *ComplexLattice::get_pointer()
{
	return store;
}

























































class ComplexLatticeSpatial
{
	public:
		ComplexLatticeSpatial(bool conjug);
		~ComplexLatticeSpatial();
		void fft();
		void fft_inv();
		void normalize(double factor);
		void operator *=(ComplexLatticeSpatial &other);
		void operator +=(ComplexLatticeSpatial &other);
		void operator =(ComplexLatticeSpatial &other);
		hipDoubleComplex *get_pointer();
	private:
		hipDoubleComplex *store;
		int length;
		bool conjugated;
		dim3 dimblock2;
		dim3 dimgrid2;
		hipfftHandle plan;
		void make_plan();
};

ComplexLatticeSpatial::ComplexLatticeSpatial(bool conjug)
{
	length=COMPONENTS*XSIZE*YSIZE*ZSIZE;
	conjugated=conjug;
	hipMalloc(&store,length*sizeof(hipDoubleComplex));
	
	
	dim3 temp_dimblock2(BLOCKSIZE,1,1);
	dim3 temp_dimgrid2(int(length/BLOCKSIZE),1,1);
	dimblock2=temp_dimblock2;
	dimgrid2=temp_dimgrid2;
	
	make_plan();
}

ComplexLatticeSpatial::~ComplexLatticeSpatial()
{
	hipFree(store);
	hipfftDestroy(plan);
}



void ComplexLatticeSpatial::make_plan()
{
	int rank = ZSIZE > 1 ? 3 : ((ZSIZE==1 && YSIZE > 1) ? 2 : ((ZSIZE==1 && YSIZE==1 && XSIZE>1) ? 1 : 0));
	int n[rank];

	if (rank == 1)
	{
		n[0] = XSIZE;
	}
	else if (rank == 2)
	{
		n[0] = XSIZE;
		n[1] = YSIZE;
	}
	else if (rank == 3)
	{
		n[0] = XSIZE;
		n[1] = YSIZE;
		n[2] = ZSIZE;
	}
	hipfftPlanMany(&plan, rank, n, NULL, 0, 0, NULL, 0, 0, HIPFFT_Z2Z, COMPONENTS);
}

void ComplexLatticeSpatial::fft()
{
	if(conjugated)
	{
		hipfftExecZ2Z(plan, store, store, HIPFFT_FORWARD);
	}
	else
	{
		hipfftExecZ2Z(plan, store, store, HIPFFT_BACKWARD);
	}
}

void ComplexLatticeSpatial::fft_inv()
{
	if(conjugated)
	{
		hipfftExecZ2Z(plan, store, store, HIPFFT_BACKWARD);
	}
	else
	{
		hipfftExecZ2Z(plan, store, store, HIPFFT_FORWARD);
	}
}

void ComplexLatticeSpatial::normalize(double factor)
{
	hipDeviceSynchronize();
	normalization<<<dimgrid2,dimblock2>>>(store, factor);
	hipDeviceSynchronize();
}


void ComplexLatticeSpatial::operator +=(ComplexLatticeSpatial &other)
{
	hipDeviceSynchronize();
	summation<<<dimgrid2,dimblock2>>>(store, other.store);
	hipDeviceSynchronize();
}

void ComplexLatticeSpatial::operator *=(ComplexLatticeSpatial &other)
{
	hipDeviceSynchronize();
	multiplication<<<dimgrid2,dimblock2>>>(store, other.store);
	hipDeviceSynchronize();
}

void ComplexLatticeSpatial::operator =(ComplexLatticeSpatial &other)
{
	hipMemcpy(store,other.store,length*sizeof(hipDoubleComplex) ,hipMemcpyDeviceToDevice);
}


hipDoubleComplex *ComplexLatticeSpatial::get_pointer()
{
	return store;
}





























class ComplexLatticeHost
{
	public:
		ComplexLatticeHost();
		~ComplexLatticeHost();
		void copy_from_device(ComplexLattice *device_lattice);
		void copy_to_device(ComplexLattice *device_lattice);
		complex <double> *get_pointer();
		int get_length();
	private:
		complex <double> *store;
		int length;	
};



ComplexLatticeHost::ComplexLatticeHost()
{
	length=COMPONENTS*XSIZE*YSIZE*ZSIZE*TAUSIZE;

	store=new complex <double> [length];
}

ComplexLatticeHost::~ComplexLatticeHost()
{
	delete [] store;
}


void ComplexLatticeHost::copy_from_device(ComplexLattice *device_lattice)
{
	hipMemcpy(store,device_lattice->get_pointer(),2*length*sizeof(double),hipMemcpyDeviceToHost);
}

void ComplexLatticeHost::copy_to_device(ComplexLattice *device_lattice)
{
	hipMemcpy(device_lattice->get_pointer(),store,2*length*sizeof(double),hipMemcpyHostToDevice);
}

complex <double> *ComplexLatticeHost::get_pointer()
{
	return store;
}

int ComplexLatticeHost::get_length()
{
	return length;
}
