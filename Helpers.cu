__device__ inline int my_map(int x, int m)
{
	if(x>=0&&x<m)
	{
		return x;
	}
	else if(x<0)
	{
		return x+m;
	}
	else
	{
		return x-m;
	}
}

__device__ inline double mysquare(double x)
{
	return x*x;
}

__device__ inline double mycube(double x)
{
	return x*x*x;
}

inline double mysquare_h(double x)
{
	return x*x;
}

inline double mycube_h(double x)
{
	return x*x*x;
}


void set_device_variable(double *devptr,double value)
{
	double temp=value;
	hipMemcpy(devptr, &temp, sizeof(double), hipMemcpyHostToDevice);
}

void set_multiple_device_variables(double *devptr,double value, int number)
{
	for(int i=0; i<number; i++)
	{
		set_device_variable(devptr+i,value);
	}
}	

double get_device_variable(double *devptr)
{
	double temp;
	hipMemcpy(&temp, devptr, sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}


__device__ hipDoubleComplex  operator+(hipDoubleComplex a, hipDoubleComplex b) { return hipCadd(a,b); }
__device__ hipDoubleComplex  operator-(hipDoubleComplex a, hipDoubleComplex b) { return hipCsub(a,b); }
__device__ hipDoubleComplex  operator*(hipDoubleComplex a, hipDoubleComplex b) { return hipCmul(a,b); }
__device__ hipDoubleComplex  operator/(hipDoubleComplex a, hipDoubleComplex b) { return hipCdiv(a,b); }

__device__ hipDoubleComplex  operator*(double a, hipDoubleComplex b) { return make_hipDoubleComplex(a*b.x,a*b.y); }
